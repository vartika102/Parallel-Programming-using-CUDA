#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include<iostream>
#include<conio.h>

#include<time.h>

using namespace std;

/*Kernel Function:
    Variables passed to the Kernel:

�	darr � Stores the input matrix.
�	dqueu � Stores the weight calculated for each vertex.
�	ddept � Stores the depth of each vertex.
�	dvis � Initially stores the value for visited vertices and later the final output path.
�	st- Stores the starting vertex.
�	nw � Stores the total no. of vertices.
�	d � Stores the value of maximum depth.
*/

__global__ void myKernel(int *darr, int *dqueu, int *ddept, int *dvis, int st, int nw, int *d)
{
    int x = threadIdx.x;
	int y = blockIdx.x;
	ddept[st] = 0;//updating the depth of the starting vertex with 0.
	dqueu[st] = 0;//Initialising the weight �dqueu� for the starting vertex with 0.
	dvis[st] = 1;//Marking the dvis for starting vertex as visited.
	
	d[0]=0;//Initialising the maximum depth with 0.

	//Calculating depth, weight(queue), dvis of each vertex and maximum depth
	while(ddept[x]==-1)
	{
	    if(darr[y*nw +x] == 1)
	    {
	        if(ddept[x] == -1)
	        {
	            if(ddept[y] != -1)
	            {
	                dvis[x] = 1;
		            ddept[x] = ddept[y]+1;
		            dqueu[x] = dqueu[y]*10 + x;
		            if(d[0]<ddept[x])
	            	    d[0]=ddept[x];
	            }
	        }
	    }
	    __syncthreads();
	    if(darr[y*nw +x] == 1)
	    {
	        if(ddept[y] == -1)
	        {
	            if(ddept[x] != -1)
	            {
	                dvis[y] = 1;
		            ddept[y] = ddept[x]+1;
		            dqueu[y] = dqueu[x]*10 + y;
		            if(d[0]<ddept[y])
	                	  d[0]=ddept[y];
	            }
	        }
	    }
        __syncthreads();
	}
	__syncthreads();

	//Updating weight of vertices having more than one parent vertices.
	for(int i=0; i<d[0]; i++)
	if(darr[y*nw+x] == 1 && ddept[x] != ddept[y])
	{
	    if(ddept[x]>ddept[y] && dqueu[y]<dqueu[x]/10)
	    {
	        if(x<10)
	            dqueu[x]=dqueu[y]*10 + x;
	        else if(x<100)
	            dqueu[x]=dqueu[y]*100 + x;
	        else if(x<1000)
           	    dqueu[x]=dqueu[y]*1000 + x;
	    }
	    __syncthreads();
	    if(ddept[y] > ddept[x] && dqueu[x] < dqueu[y]/10)
	    {
	        if(y<10)
	            dqueu[y] = dqueu[x]*10 + y;
	        else if(y<100)
	            dqueu[y] = dqueu[x]*100 + y;
	        else if(y<1000)
	            dqueu[y] = dqueu[x]*1000 + y;
	    }
	    __syncthreads();
	}
	__syncthreads();

	//Arranging the vertices in increasing order of their weights
	int n1=0;
	x = threadIdx.x;
	for(int i=0; i<nw; i++)
	    if(dqueu[i]<dqueu[x])
	        n1++;
	dvis[n1]=x;
	__syncthreads();

}


int main(void)
{
    /*
    Firstly, all the required variables are declared,

    Variables on the Host:
    �	array- Stores a matrix of size n* n (where n is the total no. of vertices) on Host, which displays the connections between the different vertices i.e. element has a value 1 if its row no. and column no. are connected, else stores 0.
    �	queue � n dimensional vector that stores the weights alotted to each vertex as they are transversed by the threads.
    �	depth � n dimensional vector that stores the depth or heirachy level of each vertex.
    �	vis- n dimensional vector whose all elements are initialised to 0 at first and then , after the kernel launch , updated with different values. It is basically a vector which stores the info whether a particular vertex is visited or not and later it is overwritten by the output.
    �	dmax � Stores the value of maximum depth.
    �	start � Stores the starting vertex.
    �	n � Stores the total no. of vertices.

    Variables on the Device:
    �	darray � n*n matrix to store the values of matrix �array� on Device.
    �	dqueue- n dimensional vector to store values for �queue� on Device.
    �	ddepth � n dimensional vector to store the values for �depth� on Device.
    �	dvist � n dimensional vector to store the values for vis on Device.
    �	dd � Stores the value of maximum depth.

    */

	//Declaration of variables
    int *array, *queue, *depth, *vis, *q, *dmax;
	int *darray, *dqueue, *ddepth, *dvist, *dque, *dd;
	int n, start;
	
	//To input the total no. of vertices
	cout<<"Enter the no. of vertices: ";
	cin>>n;

	//memory allocation on Host
	array = (int*)malloc(sizeof(int)*n*n);
	queue = (int*)malloc(sizeof(int)*n);
	depth = (int*)malloc(sizeof(int)*n);
	vis = (int*)malloc(sizeof(int)*n);
	q = (int*)malloc(sizeof(int)*n);
	dmax = (int*)malloc(sizeof(int)*2);
	
	//Initialising all the elements of the input matrix to 0
	for(int i=0; i<n; i++)
	for(int j=0; j<n; j++)
	{
	    array[i*n+j]=0;
	}

	//Input to the vertices and their connections
	int c;
	for(int i=0; i<n; i++)
	{
	    int x;
		cout<<"Enter the no. of vertices connected with "<<i<<": ";
		cin>>x;
		cout<<"Enter the vertices: ";
		for(int j=0; j<x; j++)
		{
		    cin>>c;
			array[i*n+c]=1;
		}
	}

	//Displaying the matrix representing the connected vertices
	for(int i=0; i<n; i++)
	{
	    for(int j=0; j<n; j++)
	        cout<<array[i*n+j]<<" ";
	    cout<<endl;
	}

	//Input to the starting vertex
	cout<<"Enter the starting vertex: ";
	cin>>start;

	//memory allocation on Device
	hipMalloc((void**)&darray,sizeof(int)*n*n);
	hipMalloc((void**)&dqueue,sizeof(int)*n);
	hipMalloc((void**)&dvist,sizeof(int)*n);
	hipMalloc((void**)&ddepth,sizeof(int)*n);
	hipMalloc((void**)&dd,sizeof(int)*2);
	hipMalloc((void**)&dque,sizeof(int)*n);

	//Initialising the vector dvist(vector to store the visited  to 0 and ddepth to -1
	hipMemset(dvist,0,sizeof(int)*n);
	hipMemset(ddepth,-1,sizeof(int)*n);

	//Copying the input matrix from Host to Device.
	hipMemcpy(darray,array,sizeof(int)*n*n,hipMemcpyHostToDevice);

	//Kernel Launch
	myKernel<<<n, n>>>(darray, dqueue, ddepth, dvist, start, n, dd);
	hipDeviceSynchronize();
	
	//Copying the updated values from Device to Host.
	hipMemcpy(array,darray,sizeof(int)*n*n,hipMemcpyDeviceToHost);
	hipMemcpy(queue,dqueue,sizeof(int)*n,hipMemcpyDeviceToHost);
	hipMemcpy(depth,ddepth,sizeof(int)*n,hipMemcpyDeviceToHost);
	hipMemcpy(vis,dvist,sizeof(int)*n,hipMemcpyDeviceToHost);
	hipMemcpy(dmax,dd,sizeof(int)*2,hipMemcpyDeviceToHost);

	//Displaying the output
	for(int i=0; i<n; i++)
	{
	    cout<<i<<" "<<depth[i]<<" ";
	    cout<<queue[i]<<" ";
	    cout<<endl;
	}
	cout<<"Maximum Depth: "<<dmax[0]<<endl;
	 
	cout<<"The Result of BFS: "; 
	for(int i=0; i<n; i++)
	    cout<<vis[i]<<" ";

   //Freeing the memory.
	free(array);
	free(queue);
	free(depth);
	free(vis);
	free(dmax);
	hipFree(darray);
	hipFree(dqueue);
	hipFree(ddepth);
	hipFree(dvist);
	hipFree(dd);

	getch();
}