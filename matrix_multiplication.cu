#include "hip/hip_runtime.h"
#include<iostream>
#include<conio.h>
#include"hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime_api.h>

using namespace std;

/*Kernel Function: All the 3 matrices along with the width(no. of elements in each row of 1st matrix)
 are recieved as arguments for the function. Each thread calculates one element of the product
 matrix i.e. (i*width+j)th thread calculates the ith row and jth column element of the product
 matrix by multiplying each element  of ith row of the 1st matrix to the corresponding elements 
 of the jth column of 2nd matrix.*/

__global__ void mulKernel(float *A1, float *B1, float *C1, int width)
{
    int row = blockIdx.x;
	int col = threadIdx.x;
	if(row<width && col<width)
	{
	    C1[row*width + col] = 0; //initialising element to 0.

	    /*To calculating the sum of the product of the corresponding elements of 
	      the (row)th row of 1st  array and (col)th column of the 2nd array inorder to 
          get the element of the (row)th row and (col)th column of the product 
          matrix*/

	    for(int i=0; i<width; i++) 
	        C1[row*width + col] += A1[row*width + i]* B1[i*width + col];
	}
}

int main(void)
{
   //declaration of Host and Device variables
   float *A, *B, *C;
   float *a, *b, *c;
   int w, am, an, bm, bn;

   //Input the dimensions of the input array
   cout<<"Enter the dimensions of the 1st array:";
   cin>>am>>an;
   cout<<"Enter the dimensions of the 2nd array:";
   cin>>bm>>bn;
   w = an; //width of the product matrix

   //memory allocation on Host Memory
   A = (float*)malloc(sizeof(float)*am*an);
   B = (float*)malloc(sizeof(float)*bm*bn);
   C = (float*)malloc(sizeof(float)*am*bn);

   //memory allocation on Device Memory
   hipMalloc((void**)&a,sizeof(float)*am*an);
   hipMalloc((void**)&b,sizeof(float)*bm*bn);
   hipMalloc((void**)&c,sizeof(float)*am*bn);

   //Input to 1st array in vector form
   cout<<"Enter the 1st array:"<<endl;
   for(int i=0; i<an*am; i++)
       cin>>A[i];

   //Input to 2nd array in vector form
   cout<<"Enter the 2nd array:"<<endl;
   for(int i=0; i<bn*bm; i++)
       cin>>B[i];

   //Copying input matrices from Host to Device
   hipMemcpy(a,A,sizeof(float)*am*an,hipMemcpyHostToDevice);
   hipMemcpy(b,B,sizeof(float)*bm*bn,hipMemcpyHostToDevice);

   /*Kernel Launch: here �am� is the no. of rows of the 1st matrix and �bn� is the no. 
     of columns of the  2nd matrix, and hence the dimensions of the product 
     matrix. Each Block corresponds to a row in the product matrix and each thread 
     corresponds to each element of the product matrix. Moreover, the 2 input 
     matrices, the product matrix and the width are passed as vector arguments to 
     the Kernel Function.*/

   mulKernel<<<am,bn>>>(a,b,c,w);
   hipDeviceSynchronize();

   //Copying product matrix from Device to Host
   hipMemcpy(C,c,sizeof(float)*am*bn,hipMemcpyDeviceToHost);

   //Output the product matrix
   cout<<"resultant array:"<<endl;
   for(int i=0; i<am; i++)
   {
      for(int j=0; j<bn; j++)
         cout<<C[i*am + j];
     cout<<endl;
   }

   getch();
   return 0;
}