#include "hip/hip_runtime.h"
#include<iostream>
#include<conio.h>
#include<stdio.h>
#include"hip/hip_runtime.h"

#include<hip/hip_runtime_api.h>

using namespace std;

/*Kernel function: It takes the 3 matrices and their size as arguments. Each thread
    calculates an element of resultant matrix.*/

__global__ void addKernel(float *a1, float *b1, int n1, float *c1)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	/*finding the sum of 2 corresponding elements from the 2 matrices represented
       by a thread*/
	if(i < n1)
	  c1[i] = a1[i] + b1[i];//sum of corresponding elements
}

int main(void)
{
    float *A, *B, *C; //Variables for matrices in Host memory
	float *a, *b, *c; //Variables for matrices in Device memory
	int n, m; //variables for no of rows and columns of the matrices

	cout<<"Enter the dimensions of the array:";
	cin>>m>>n; //To enter the dimensions of matrices

	int x = sizeof(float) * n * m; //to find the size of the matrices in bytes

	//Memory allocation for host variables
	A = (float*)malloc(x);
	B = (float*)malloc(x);
    C = (float*)malloc(x);
	
	int y = m * n; //Calculation of total no. of elements in each matrix
	
	//Memory allocation for device variables
	hipMalloc((void**)&a, x);
	hipMalloc((void**)&b, x);
	hipMalloc((void**)&c, x);
		
	//Input to the 1st array
	cout<<"Enter the elements in the 1st array:"<<endl;
	for(int i=0; i<y; i++)
	{
	  cin>>A[i];
	}
	
	//Input to th 2nd array
	cout<<"Enter the elements in the 2nd array:"<<endl;
	for(int i=0; i<y; i++)
	{
	   cin>>B[i];
	}

	//Copying the arrays from Host to Device
	hipMemcpy(a, A, x, hipMemcpyHostToDevice);
	hipMemcpy(b, B, x, hipMemcpyHostToDevice);
	
	/*Kernel Launch: Each matrix has dimensions m*n and so a total of m blocks 
        and n threads per block are alotted in a way that each thread corresponds to 
        an element in the resultant matrix. Moreover, all the matrices are passed as 
        arguments to the kernel along with the total no. of elements in each 
        matrix.*/

	addKernel<<<m, n>>>(a, b, y, c);
	hipDeviceSynchronize();

	//Copying the resultant array from Host to Device
	hipMemcpy(C, c, x, hipMemcpyDeviceToHost);

	//Output the resultant array
	cout<<"the sum array:"<<endl;
	for(int i=0; i<y; i++)
	{
	   cout<<C[i]<<endl;
	}

	getch();

	//freeing the memory
	free(A);
    free(B);
    free(C);
	hipFree(a);
	hipFree(b);
	hipFree(c);
	return 0;
}